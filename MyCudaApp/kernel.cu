﻿
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include "exemplos.cuh"


int main()
{
	hipDeviceReset();
	execute_matrix_mult_gpu(1000);
	execute_matrix_mult_cpu(1000);
	
	return 0;
}




