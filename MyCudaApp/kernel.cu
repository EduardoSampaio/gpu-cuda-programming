﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>


#define RADIUS 3
#define GRIDSIZE 2
#define BLOCKSIZE 512
#define THREAD_PER_BLOCK = 32;

typedef unsigned long int NUMBER;
const NUMBER N = 99999999;

__global__ void stencil_1d(NUMBER* in, NUMBER* out) {
	__shared__ NUMBER temp[BLOCKSIZE + 2 * RADIUS];
	NUMBER gindex = threadIdx.x + blockIdx.x * blockDim.x;
	NUMBER lindex = threadIdx.x + RADIUS;

	temp[lindex] = in[gindex];

	if (threadIdx.x < RADIUS) {
		temp[lindex - RADIUS] = in[lindex - RADIUS];
		temp[lindex + BLOCKSIZE] = in[gindex + BLOCKSIZE];
	}

	__syncthreads();

	NUMBER result = 0;
	for (NUMBER offset = -RADIUS; offset <= RADIUS; offset++) {
		result += temp[lindex + offset];
	}

	out[gindex] = result;
}

void print_stencil(int* v) {
	printf("[");
	for (int i = 0; i < N; i++) {
		if (i != N - 1) {
			printf("%d,", v[i]);
		}
		else {
			printf("%d", v[i]);
		}
	}
	printf("]");
}

int main()
{
	NUMBER* h_in, * h_out, * d_in, * d_out;

	h_in = (NUMBER*)malloc(N * sizeof(NUMBER));
	h_out = (NUMBER*)malloc(N * sizeof(NUMBER));

	for (NUMBER i = 0; i < N; i++) {
		h_in[i] = 1;
	}

	hipEvent_t start = hipEvent_t();
	hipEvent_t stop = hipEvent_t();
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMalloc((void**)&d_in, N * sizeof(NUMBER));
	hipMalloc((void**)&d_out, N * sizeof(NUMBER));

	hipMemcpy(d_in, h_in, N * sizeof(NUMBER), hipMemcpyHostToDevice);
	hipMemcpy(d_out, h_out, N * sizeof(NUMBER), hipMemcpyHostToDevice);

	hipEventRecord(start,0);

	stencil_1d<<<GRIDSIZE, BLOCKSIZE>>>(d_in, d_out);
	hipDeviceSynchronize();
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("Time elapsed on CPU: %f ms.\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}





