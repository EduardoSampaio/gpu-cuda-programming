#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

__global__ void sum(int* a, int* b, int* c) {
	*c = *a + *b;
}

__global__ void vector_sum(int* a, int* b, int* c, int n) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (n > index) {
		c[index] = a[index] + b[index];
	}
}

void exemplo01() {
	int a, b, c;
	int* d_a, * d_b, * d_c;

	hipDeviceReset();

	a = 10;
	b = 20;

	hipMalloc((void**)&d_a, sizeof(int));
	hipMalloc((void**)&d_b, sizeof(int));
	hipMalloc((void**)&d_c, sizeof(int));

	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

	sum << <1, 1 >> > (d_a, d_b, d_c);
	hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

	printf("Resultado %d", c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}


int main()
{
	hipDeviceReset();
	const int THREAD_PER_BLOCK = 32;
	const int N = 2048;
	int* a, * b, * c;
	int* d_a, * d_b, * d_c;
	int size = N * sizeof(int);

	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);


	for (int i = 0; i < N; ++i) {
		a[i] = rand() % 100;
		b[i] = rand() % 100;
	}

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	hipMemset((void*)&d_a, 0, size);
	hipMemset((void*)&d_b, 0, size);
	hipMemset((void*)&d_c, 0, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	vector_sum << <N / THREAD_PER_BLOCK, THREAD_PER_BLOCK >> > (d_a, d_b, d_c, N);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);


	for (int i = 0; i < N; i++) {
		printf("VETOR[%d]=%d\n", i, c[i]);
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	free(a);
	free(b);
	free(c);


	return 0;
}




